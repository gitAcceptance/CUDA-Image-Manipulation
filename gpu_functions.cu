#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime_api.h> 
#include <>
#include "hip/hip_runtime_api.h"

using namespace std;

__global__
void gaussian_blur(const unsigned char* const inputChannel,
	unsigned char* const outputChannel,
	int numRows, int numCols,
	const float* const filter, const int filterWidth)
{
	// TODO

	// NOTE: Be sure to compute any intermediate results in floating point
	// before storing the final result as unsigned char.

	// NOTE: Be careful not to try to access memory that is outside the bounds of
	// the image. You'll want code that performs the following check before accessing
	// GPU memory:
	//
	// if ( absolute_image_position_x >= numCols ||
	//      absolute_image_position_y >= numRows )
	// {
	//     return;
	// }

	// NOTE: If a thread's absolute position 2D position is within the image, but some of
	// its neighbors are outside the image, then you will need to be extra careful. Instead
	// of trying to read such a neighbor value from GPU memory (which won't work because
	// the value is out of bounds), you should explicitly clamp the neighbor values you read
	// to be within the bounds of the image. If this is not clear to you, then please refer
	// to sequential reference solution for the exact clamping semantics you should follow.

	const int absolute_pos_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int absolute_pos_y = blockIdx.y * blockDim.y + threadIdx.y;
	const int index = absolute_pos_y * numCols + absolute_pos_x;

	if (absolute_pos_x >= numCols || absolute_pos_y >= numRows)
		return;

	float new_color = 0.0f;

	for (int filter_y = 0; filter_y < filterWidth; filter_y++) {
		for (int filter_x = 0; filter_x < filterWidth; filter_x++) {

			int local_x = absolute_pos_x + filter_x - filterWidth / 2;
			int local_y = absolute_pos_y + filter_y - filterWidth / 2;

			local_x = min(max(local_x, 0), numCols - 1); // This is how we clamp the values when we're working on an edge pixel
			local_y = min(max(local_y, 0), numRows - 1);

			float filter_factor = filter[filter_y * filterWidth + filter_x]; // gotta get our filter values

			new_color += filter_factor * static_cast<float>(inputChannel[local_y * numCols + local_x]);

		}
	}

	outputChannel[index] = new_color;


}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
	int numRows,
	int numCols,
	unsigned char* const redChannel,
	unsigned char* const greenChannel,
	unsigned char* const blueChannel)
{
	// TODO
	//
	// NOTE: Be careful not to try to access memory that is outside the bounds of
	// the image. You'll want code that performs the following check before accessing
	// GPU memory:
	//
	// if ( absolute_image_position_x >= numCols ||
	//      absolute_image_position_y >= numRows )
	// {
	//     return;
	// }

	const int absolute_pos_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int absolute_pos_y = blockIdx.y * blockDim.y + threadIdx.y;
	const int m = absolute_pos_y * numCols + absolute_pos_x;

	if (absolute_pos_x >= numCols || absolute_pos_y >= numRows)
		return;

	redChannel[m] = inputImageRGBA[m].x;
	greenChannel[m] = inputImageRGBA[m].y;
	blueChannel[m] = inputImageRGBA[m].z;



}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
	const unsigned char* const greenChannel,
	const unsigned char* const blueChannel,
	uchar4* const outputImageRGBA,
	int numRows,
	int numCols)
{
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	//make sure we don't try and access memory outside the image
	//by having any threads mapped there return early
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	unsigned char red = redChannel[thread_1D_pos];
	unsigned char green = greenChannel[thread_1D_pos];
	unsigned char blue = blueChannel[thread_1D_pos];

	//Alpha should be 255 for no transparency
	uchar4 outputPixel = make_uchar4(red, green, blue, 255);

	outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
	const float* const h_filter, const size_t filterWidth)
{

	//allocate memory for the three different channels
	//original
	checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRowsImage * numColsImage));
	checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
	checkCudaErrors(hipMalloc(&d_blue, sizeof(unsigned char) * numRowsImage * numColsImage));

	//TODO:
	//Allocate memory for the filter on the GPU
	//Use the pointer d_filter that we have already declared for you
	//You need to allocate memory for the filter with hipMalloc
	//be sure to use checkCudaErrors like the above examples to
	//be able to tell if anything goes wrong
	//IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc
	checkCudaErrors(hipMalloc(&d_filter, sizeof(float) * filterWidth * filterWidth));

	//TODO:
	//Copy the filter on the host (h_filter) to the memory you just allocated
	//on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
	//Remember to use checkCudaErrors!
	checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));

}

void gpu_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
	uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
	unsigned char *d_redBlurred,
	unsigned char *d_greenBlurred,
	unsigned char *d_blueBlurred,
	const int filterWidth)
{
	//TODO: Set reasonable block size (i.e., number of threads per block)
	const dim3 threadsPerBlock(32, 32, 1); // aka blockSize

										   //TODO:
										   //Compute correct grid size (i.e., number of blocks per kernel launch)
										   //from the image size and and block size.
	const dim3 numberOfBlocks(numCols / threadsPerBlock.x + 1, numRows / threadsPerBlock.y + 1, 1); // aka gridSize

	const dim3 gridSize = numberOfBlocks;
	const dim3 blockSize = threadsPerBlock;

	//TODO: Launch a kernel for separating the RGBA image into different color channels
	// void separateChannels(const uchar4* const inputImageRGBA, int numRows, int numCols, unsigned char* const redChannel, unsigned char* const greenChannel, unsigned char* const blueChannel)

	separateChannels<<<gridSize, blockSize >>>(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);

	// Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
	// launching your kernel to make sure that you didn't make any mistakes.
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	//TODO: Call your convolution kernel here 3 times, once for each color channel.

	// Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
	// launching your kernel to make sure that you didn't make any mistakes.

	gaussian_blur<<<gridSize, blockSize >>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);

	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	gaussian_blur<<<gridSize, blockSize >>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);

	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	gaussian_blur<<<gridSize, blockSize >>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);

	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	// Now we recombine your results. We take care of launching this kernel for you.
	//
	// NOTE: This kernel launch depends on the gridSize and blockSize variables,
	// which you must set yourself.
	recombineChannels<<<gridSize, blockSize >>>(d_redBlurred,
		d_greenBlurred,
		d_blueBlurred,
		d_outputImageRGBA,
		numRows,
		numCols);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
	checkCudaErrors(hipFree(d_red));
	checkCudaErrors(hipFree(d_green));
	checkCudaErrors(hipFree(d_blue));
}
